#include "hip/hip_runtime.h"
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
// Libreries for use CUDA FRAMEWORK	to run in parallel an implementation of Mergesort algorithm using c language
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#define N1 1
#define N10 10
#define N100 100
#define N1000 1000

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//		Function to combine the sublists generated. We use the flag __device__ because it is called using the function CudaMergeSort which is working on the gpu. 
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__device__ void CudaMerge(int* values, int* results, int l, int r, int u)
{
	int i, j, k;
	i = l; j = r; k = l;
	while (i < r && j < u) {
		if (values[i] <= values[j]) { results[k] = values[i]; i++; }
		else { results[k] = values[j]; j++; }
		k++;
	}

	while (i < r) {
		results[k] = values[i]; i++; k++;
	}

	while (j < u) {
		results[k] = values[j]; j++; k++;
	}
	for (k = l; k < u; k++) {
		values[k] = results[k];
	}
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//		Function to generate the sublists of the array to sort. It use flag __global__ because the function is called from the main. It is also call Kernel and use a specific call. 
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ static void CudaMergeSort(int * values, int* results, int dim)
{
	extern __shared__ int shared[];



	const unsigned int tid = threadIdx.x;
	int k, u, i;
	shared[tid] = values[tid];


	__syncthreads();
	k = 1;
	while (k <= dim)
	{
		i = 0;
		while (i + k < dim)
		{
			u = i + k * 2;;
			if (u > dim)
			{
				u = dim + 1;
			}
			CudaMerge(shared, results, i, i + k, u);
			i = i + k * 2;
		}
		k = k * 2;

		__syncthreads();
	}

	values[tid] = shared[tid];
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
// Declaration of the function MergeSort implemented in C, to study the different times of execution
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
void mergeSort(int arr[], int p, int q);


int main(int argc, char** argv)
{	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		VARIABLES USED DURING EXECUTION
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		TIMING CUDA OPERATIONS USING EVENTS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	float elapsed1, elapsed10, elapsed100, elapsed1000 = 0;
	hipEvent_t start1, stop1, start10, stop10, start100, stop100, start1000, stop1000;
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		CREATING THE EVENTS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	hipEventCreate(&start1);
	hipEventCreate(&start10);
	hipEventCreate(&start100);
	hipEventCreate(&start1000);
	hipEventCreate(&stop1);
	hipEventCreate(&stop10);
	hipEventCreate(&stop100);
	hipEventCreate(&stop1000);
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAYS TO STUDY
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	int values1[N1], values2[N10], values3[N100], values4[N1000];
	// Creating copies for the study in sequential. 
	int values1s[N1], values2s[N10], values3s[N100], values4s[N1000];
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		POINTERS TO USE ARRAYS IN GPU OPERATIONS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	int* dvalues, *results;
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//																									MAIN CODE
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	printf("\n\t\t ---------------------- PARALLEL IMPLEMENTATION ---------------------- ");
	//		THE PROGRAM CREATES ARRAYS FOR 1, 10, 100 AND 1000 ELEMENTS WITH RANDOM VALUES
	printf("\nCreating vector for N = 1, 10, 100 and 1000 with random values.\n");
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 1 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	// Creating the array
	printf("\nElements for N = 1:\n");
	for (int i = 0; i < N1; i++)
	{
		values1[i] = rand();
		values1s[i] = values1[i];
		printf("%d ", values1[i]);
	}
	printf("\n");
	//Start monitorizing of cuda operations
	hipEventRecord(start1, 0);
	//Generation cuda variables ables to work and copying the variables from host to device
	hipMalloc((void**)&dvalues, sizeof(int) * N1);
	hipMemcpy(dvalues, values1, sizeof(int) * N1, hipMemcpyHostToDevice);
	hipMalloc((void**)&results, sizeof(int) * N1);
	hipMemcpy(results, values1, sizeof(int)* N1, hipMemcpyHostToDevice);
	//Calling algorithm MergeSort
	CudaMergeSort << <1, N1, sizeof(int) * N1 * 2 >> > (dvalues, results, N1);
	// Freeing memory space used
	hipFree(dvalues);
	hipMemcpy(values1, results, sizeof(int)*N1, hipMemcpyDeviceToHost);
	hipFree(results);
	//Stopping time monitoring
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	//Calculating the total time of execution
	hipEventElapsedTime(&elapsed1, start1, stop1);
	// Freeing the events created before
	hipEventDestroy(start1);
	hipEventDestroy(stop1);
	//Showing sorted elements
	printf("\nSorted Elements:\n");
	for (int i = 0; i < N1; i++)
	{
		printf("%d ", values1[i]);
	}
	//Showing the time of execution
	printf("\n\t||| The elapsed time in gpu was %.2f ms |||", elapsed1);
	printf("\n");
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 10 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	// Creating the array
	printf("\nElements for N = 10:\n");
	for (int i = 0; i < N10; i++)
	{
		values2[i] = rand();
		values2s[i] = values2[i];
		printf("%d ", values2[i]);
	}
	printf("\n");
	//Start monitorizing of cuda operations
	hipEventRecord(start10, 0);
	//Generation cuda variables ables to work and copying the variables from host to device
	hipMalloc((void**)&dvalues, sizeof(int) * N10);
	hipMemcpy(dvalues, values2, sizeof(int) * N10, hipMemcpyHostToDevice);
	hipMalloc((void**)&results, sizeof(int) * N10);
	hipMemcpy(results, values2, sizeof(int)* N10, hipMemcpyHostToDevice);
	//Calling algorithm MergeSort
	CudaMergeSort << <1, N10, sizeof(int) * N10 * 2 >> > (dvalues, results, N10);
	// Freeing memory space used
	hipFree(dvalues);
	hipMemcpy(values2, results, sizeof(int)*N10, hipMemcpyDeviceToHost);
	hipFree(results);
	//Stopping time monitoring
	hipEventRecord(stop10, 0);
	hipEventSynchronize(stop10);
	//Calculating the total time of execution
	hipEventElapsedTime(&elapsed10, start10, stop10);
	// Freeing the events created before
	hipEventDestroy(start10);
	hipEventDestroy(stop10);
	//Showing sorted elements
	printf("\nSorted Elements:\n");
	for (int i = 0; i < N10; i++)
	{
		printf("%d ", values2[i]);
	}
	//Showing the time of execution
	printf("\n\t||| The elapsed time in gpu was %.2f ms |||", elapsed10);
	printf("\n");

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 100 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	// Creating the array
	printf("\nElements for N = 100:\n");
	for (int i = 0; i < N100; i++)
	{
		values3[i] = rand();
		values3s[i] = values3[i];
		printf("%d ", values3[i]);
	}
	printf("\n");
	//Start monitorizing of cuda operations
	hipEventRecord(start100, 0);
	//Generation cuda variables ables to work and copying the variables from host to device
	hipMalloc((void**)&dvalues, sizeof(int) * N100);
	hipMemcpy(dvalues, values3, sizeof(int) * N100, hipMemcpyHostToDevice);
	hipMalloc((void**)&results, sizeof(int) * N100);
	hipMemcpy(results, values3, sizeof(int)* N100, hipMemcpyHostToDevice);
	//Calling algorithm MergeSort
	CudaMergeSort << <1, N100, sizeof(int) * N100 * 2 >> > (dvalues, results, N100);
	// Freeing memory space used
	hipFree(dvalues);
	hipMemcpy(values3, results, sizeof(int)*N100, hipMemcpyDeviceToHost);
	hipFree(results);
	//Stopping time monitoring
	hipEventRecord(stop100, 0);
	hipEventSynchronize(stop100);
	//Calculating the total time of execution
	hipEventElapsedTime(&elapsed100, start100, stop100);
	// Freeing the events created before
	hipEventDestroy(start100);
	hipEventDestroy(stop100);
	//Showing sorted elements
	printf("\nSorted Elements:\n");
	for (int i = 0; i < N100; i++)
	{
		printf("%d ", values3[i]);
	}
	//Showing the time of execution
	printf("\n\t||| The elapsed time in gpu was %.2f ms |||", elapsed100);
	printf("\n");

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 1000 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	// Creating the array
	printf("\nElements for N = 1000:\n");
	for (int i = 0; i < N1000; i++)
	{
		values4[i] = rand();
		values4s[i] = values4[i];
		printf("%d ", values4[i]);
	}
	printf("\n");
	//Start monitorizing of cuda operations
	hipEventRecord(start1000, 0);
	//Generation cuda variables ables to work and copying the variables from host to device
	hipMalloc((void**)&dvalues, sizeof(int) * N1000);
	hipMemcpy(dvalues, values4, sizeof(int) * N1000, hipMemcpyHostToDevice);
	hipMalloc((void**)&results, sizeof(int) * N1000);
	hipMemcpy(results, values4, sizeof(int)* N1000, hipMemcpyHostToDevice);
	//Calling algorithm MergeSort
	CudaMergeSort << <1, N1000, sizeof(int) * N1000 * 2 >> > (dvalues, results, N1000);
	// Freeing memory space used and returning values sortered
	hipFree(dvalues);
	hipMemcpy(values4, results, sizeof(int)*N1000, hipMemcpyDeviceToHost);
	hipFree(results);
	//Stopping time monitoring
	hipEventRecord(stop1000, 0);
	hipEventSynchronize(stop1000);
	//Calculating the total time of execution
	hipEventElapsedTime(&elapsed1000, start1000, stop1000);
	// Freeing the events created before
	hipEventDestroy(start1000);
	hipEventDestroy(stop1000);
	//Showing sorted elements
	printf("\nSorted Elements:\n");
	for (int i = 0; i < N1000; i++)
	{
		printf("%d ", values4[i]);
	}
	//Showing the time of execution
	printf("\n\t||| The elapsed time in gpu was %.2f ms |||", elapsed1000);
	printf("\n");

	hipDeviceReset();
	hipDeviceReset();

	printf("\n\t\t ---------------------- SEQUENTIAL IMPLEMENTATION ---------------------- ");

	float elapsed1s, elapsed10s, elapsed100s, elapsed1000s = 0;
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 1 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Mergesort for array with one random value
	printf("\nElements for N = 1:\n");
	for (int i = 0; i < N1; i++) {
		printf("%d ", values1s[i]);
	}
	printf("\n\nSorted Elements:");
	//Calling mergesort in c
	clock_t start = clock();
	mergeSort(values1s, 0, N1);
	elapsed1s = ((((double)clock() - start) / CLOCKS_PER_SEC) * 1000000);
	printf("\n");

	for (int i = 0; i < N1; i++) {
		printf("%d ", values1s[i]);
	}
	printf("\n\t||| The elapsed time in cpu was %.2f ms |||", elapsed1s);

	printf("\n");
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 10 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Mergesort for array with one random value
	printf("\nElements for N = 10:\n");
	for (int i = 0; i < N10; i++) {
		printf("%d ", values2s[i]);
	}
	printf("\n\nSorted Elements:");
	//Calling mergesort in c
	mergeSort(values2s, 0, N10 - 1);
	elapsed10s = ((((double)clock() - start - elapsed1) / CLOCKS_PER_SEC) * 1000000);
	printf("\n");

	for (int i = 0; i < N10; i++) {
		printf("%d ", values2s[i]);
	}
	printf("\n\t||| The elapsed time in cpu was %.2f ms |||", elapsed10s);

	printf("\n");

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 100 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Mergesort for array with one random value
	printf("\nElements for N = 100:\n");
	for (int i = 0; i < N100; i++) {
		printf("%d ", values3s[i]);
	}
	printf("\n\nSorted Elements:");
	//Calling mergesort in c
	mergeSort(values3s, 0, N100 - 1);
	elapsed100s = ((((double)clock() - start - elapsed10) / CLOCKS_PER_SEC) * 1000000);
	printf("\n");

	for (int i = 0; i < N100; i++) {
		printf("%d ", values3s[i]);
	}
	printf("\n\t||| The elapsed time in cpu was %.2f ms |||", elapsed100s);

	printf("\n");

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//		ARRAY WITH 1000 ELEMENT
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	//Mergesort for array with one random value
	printf("\nElements for N = 1000:\n");
	for (int i = 0; i < N1000; i++) {
		printf("%d ", values4s[i]);
	}
	printf("\n\nSorted Elements:");
	//Calling mergesort in c
	mergeSort(values4s, 0, N1000 - 1);
	elapsed1000s = ((((double)clock() - start - elapsed100) / CLOCKS_PER_SEC) * 1000000);
	printf("\n");

	for (int i = 0; i < N1000; i++) {
		printf("%d ", values4s[i]);
	}
	printf("\n\t||| The elapsed time in cpu was %.2f ms |||", elapsed1000s);

	printf("\n");

	return 0;
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//		Function to combine the different sublists generated and create the array sortered.
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void merge(int arr[], int p, int q, int r) {

	int i, j, k;
	int n1 = q - p + 1;
	int n2 = r - q;

	//arrs temporales
	int L[n1], M[n2];

	for (int i = 0; i < n1; i++)
	{
		L[i] = arr[p + i];
	}

	for (int j = 0; j < n2; j++)
	{
		M[j] = arr[q + 1 + j];
	}

	i = 0;
	j = 0;
	k = p;

	while (i < n1 && j < n2)
	{
		if (L[i] <= M[j])
		{
			arr[k] = L[i];
			i++;
		}
		else
		{
			arr[k] = M[j];
			j++;
		}
		k++;
	}

	while (i < n1)
	{
		arr[k] = L[i];
		i++;
		k++;
	}

	while (j < n2)
	{
		arr[k] = M[j];
		j++;
		k++;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
//		Function which subdivide the array in sublist to be ordered separetly. It use recursivity and it is based in the divide and conquer paradigm. 
//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------

void mergeSort(int arr[], int p, int q) {

	if (p < q)
	{
		int mitad = (p + q) / 2;

		mergeSort(arr, p, mitad);
		mergeSort(arr, mitad + 1, q);
		merge(arr, p, mitad, q);
	}
}
